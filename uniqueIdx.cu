////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

//
// This sample illustrates the usage of CUDA events for both GPU timing and
// overlapping CPU and GPU execution.  Events are inserted into a stream
// of CUDA calls.  Since CUDA stream calls are asynchronous, the CPU can
// perform computations while GPU is executing (including DMA memcopies
// between the host and device).  CPU can query CUDA events to determine
// whether GPU has completed tasks.
//

// includes, system
#include <stdio.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>

__global__ void unique_idx_1d_grid_1d_block(int *input)
{
	int gid = threadIdx.x + blockIdx.x * blockDim.x;
	printf("gid=%d, value=%d\n",gid, input[gid]); 

}

__global__  void  unique_idx_1d_grid_2d_block(int *input)
{
      int gid = threadIdx.x + blockIdx.x * blockDim.x * blockDim.y
	         + blockDim.x *threadIdx.y;
      
      printf("gid=%d, value=%d\n",gid, input[gid]); 

}

__global__  void  unique_idx_1d_grid_3d_block(int *input)
{
      int gid = threadIdx.x + blockIdx.x * blockDim.x * blockDim.y * blockDim.z +
	        threadIdx.z * blockDim.y * blockDim.x + 
		threadIdx.y * blockDim.x; 
      
      printf("gid=%d, value=%d\n",gid, input[gid]); 

}



__global__  void  unique_idx_2d_grid_1d_block(int *input)
{
      int blockId = blockIdx.y * gridDim.x + blockIdx.x;
      int gid = blockId  * blockDim.x + threadIdx.x;
      
      printf("gid=%d, value=%d\n",gid, input[gid]); 
}


__global__  void  unique_idx_2d_grid_2d_block(int *input)
{
      int blockId = blockIdx.x  + blockIdx.y * gridDim.x;

      int gid = blockId  * (blockDim.x * blockDim.y) 
	        + threadIdx.y * blockDim.x + threadIdx.x;
      
      printf("gid=%d, value=%d\n",gid, input[gid]); 
}


__global__  void  unique_idx_2d_grid_3d_block(int *input)
{
	int blockId = blockIdx.x  + blockIdx.y * gridDim.x;
	int gid = blockId *( blockDim.x * blockDim.y * blockDim.z)
		 + threadIdx.z * (blockDim.x * blockDim.y)
		 + threadIdx.y * blockDim.x  + threadIdx.x;
        printf("gid=%d, value=%d\n",gid, input[gid]);
}




int main(void)
{
    int array_size = 256;
    int array_byte_size = sizeof(int) * array_size;
    int *h_data = (int *) malloc(array_byte_size);
    for (int i = 0; i< array_size; i++)
    {   
	 h_data[i] = (i+1) * 10;
         printf("%d ", h_data[i]);
    }
    printf("\n\n");
    int *d_data;
    hipMalloc((void **)&d_data, array_byte_size);
    hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);
   
#if 1 
    printf("\n1D grid of 1D block:\n");
    dim3 block11(4);
    dim3 grid11(2);
    unique_idx_1d_grid_1d_block<<<grid11,block11 >>>(d_data);
    printf("\n\n");
    hipDeviceSynchronize();
#endif

#if 1
    printf("\n1D grid of 2D block:\n");
    dim3 block12(4,2);
    dim3 grid12(2);
    unique_idx_1d_grid_2d_block<<<grid12,block12>>>(d_data);
    printf("\n\n");
    hipDeviceSynchronize();
#endif

#if 1
    printf("\n1D grid of 3D block:\n");
    dim3 block13(8,4,2);
    dim3 grid13(2);
    unique_idx_1d_grid_3d_block<<<grid13,block13>>>(d_data);
    printf("\n\n");
    hipDeviceSynchronize();
#endif

#if 1
    printf("\n2D grid of 1D block:\n");
    dim3 block21(4);
    dim3 grid21(4,4);
    unique_idx_2d_grid_1d_block<<<grid21,block21>>>(d_data);
    printf("\n\n");
    hipDeviceSynchronize();
#endif


 #if 1
    printf("\n2D grid of 2D block:\n");
    dim3 block22(4,4);
    dim3 grid22(4,4);
    unique_idx_2d_grid_2d_block<<<grid22,block22>>>(d_data);
    printf("\n\n");
    hipDeviceSynchronize();
#endif

   
 #if 1
    printf("\n2D grid of 3D block:\n");
    dim3 block23(2,2,2);
    dim3 grid23(4,4);
    unique_idx_2d_grid_3d_block<<<grid23,block23>>>(d_data);
    printf("\n\n");
    hipDeviceSynchronize();
#endif



    hipDeviceReset();
    hipFree(d_data);
    free(h_data); 
    return 0;
}
